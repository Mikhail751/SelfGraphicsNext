
#include <hip/hip_runtime.h>
﻿extern "C" {
	__constant__ float ToRad = 0.017453292519943295769236907684886;
	__constant__ float ToDeg = 57.295779513082320876798154814105;
	__constant__ int CountOfTrns = 512;
	__constant__ float3 trPoint1[512];
	__constant__ float3 trPoint2[512];
	__constant__ float3 trPoint3[512];
	__constant__ float3 normals[512];
	__constant__ float dRatios[512];
	__constant__ float3 colors[512];
	__global__ void resultPixel(const int2 resolution, const float3 xyz, const float3 light, const float fow, const float2 view, float3* outColors) {
		int row = blockIdx.y * blockDim.y + threadIdx.y;
		int col = blockIdx.x * blockDim.x + threadIdx.x;
		int i = col + row * resolution.x;
		float3 y;
		y.x = 255;
		y.y = 255;
		y.z = 255;
		outColors[i] = y;
		if (col >= resolution.x || row >= resolution.y) {
			return;
		}
		float3 dir;
		dir.y = col;
		dir.z = row;
		int2 halfRes;
		halfRes.x = resolution.x / 2;
		halfRes.y = resolution.y / 2;
		dir.y -= halfRes.x;
		dir.z -= halfRes.y;
		dir.z *= -1;
		dir.y /= halfRes.x;
		dir.z /= halfRes.y;
		float halfLenHor = sinf(fow * ToRad / 2);
		dir.y *= halfLenHor;
		dir.x = sqrt(1 - powf(dir.y, 2));
		float halfLenVer = halfLenHor / resolution.x * resolution.y;
		dir.z *= halfLenVer;
		float verRatio = sqrt(1 - pow(dir.z, 2));
		if (view.y != 0) {
			float2 temp;
			temp.y = verRatio * cos(view.y * ToRad) - dir.z * sin(view.y * ToRad);
			temp.x = verRatio * sin(view.y * ToRad) + dir.z * cos(view.y * ToRad);
			dir.z = temp.x;
			verRatio = temp.y;
		}
		dir.x *= verRatio;
		dir.y *= verRatio;
		if (view.x != 0) {
			float2 temp;
			temp.x = dir.x * cos(view.x * ToRad) - dir.y * sin(view.x * ToRad);
			temp.y = dir.x * sin(view.x * ToRad) + dir.y * cos(view.x * ToRad);
			dir.x = temp.x;
			dir.y = temp.y;
		}
		float3 baseColor;
		baseColor.x = 16;
		baseColor.y = 16;
		baseColor.z = 16;
		float minDist = 10000000;
		float3 out;
		out = baseColor;
		float3 newPoint;
		int skip;
		bool coled = false;
		float colRatio = 1;
		float2 pts[4];
		for (int j = 0; j < CountOfTrns; j++)
		{
			float3 abc = normals[j];
			float upper = dRatios[j] + abc.x * xyz.x + abc.y * xyz.y + abc.z * xyz.z;
			float lower = abc.x * dir.x + abc.y * dir.y + abc.z * dir.z;
			if (lower == 0 && upper == 0) {
				continue;
			}
			if (upper > 0 && lower == 0) {
				continue;
			}
			float t = -(upper / lower);
			if (t < 0) {
				continue;
			}
			newPoint.x = dir.x * t + xyz.x;
			newPoint.y = dir.y * t + xyz.y;
			newPoint.z = dir.z * t + xyz.z;
			if (abc.x != 0) {
				pts[0].x = newPoint.y;
				pts[0].y = newPoint.z;
				pts[1].x = trPoint1[j].y;
				pts[1].y = trPoint1[j].z;
				pts[2].x = trPoint2[j].y;
				pts[2].y = trPoint2[j].z;
				pts[3].x = trPoint3[j].y;
				pts[3].y = trPoint3[j].z;
			}
			else if (abc.y != 0) {
				pts[0].x = newPoint.x;
				pts[0].y = newPoint.z;
				pts[1].x = trPoint1[j].x;
				pts[1].y = trPoint1[j].z;
				pts[2].x = trPoint2[j].x;
				pts[2].y = trPoint2[j].z;
				pts[3].x = trPoint3[j].x;
				pts[3].y = trPoint3[j].z;
			}
			else {
				pts[0].x = newPoint.x;
				pts[0].y = newPoint.y;
				pts[1].x = trPoint1[j].x;
				pts[1].y = trPoint1[j].y;
				pts[2].x = trPoint2[j].x;
				pts[2].y = trPoint2[j].y;
				pts[3].x = trPoint3[j].x;
				pts[3].y = trPoint3[j].y;
			}
			float a;
			a = (pts[1].x - pts[0].x) * (pts[2].y - pts[1].y) - (pts[2].x - pts[1].x) * (pts[1].y - pts[0].y);
			float b;
			b = (pts[2].x - pts[0].x) * (pts[3].y - pts[2].y) - (pts[3].x - pts[2].x) * (pts[2].y - pts[0].y);
			float c;
			c = (pts[3].x - pts[0].x) * (pts[1].y - pts[3].y) - (pts[1].x - pts[3].x) * (pts[3].y - pts[0].y);
			bool isIn = (a >= 0 && b >= 0 && c >= 0) || (a <= 0 && b <= 0 && c <= 0);
			if (isIn) {

				float newLen = sqrt(pow(newPoint.x - xyz.x, 2) + pow(newPoint.y - xyz.y, 2) + pow(newPoint.z - xyz.z, 2));
				bool around = newLen < minDist;
				if (around) {
					minDist = newLen;
					colRatio = (light.x - newPoint.x) * abc.x
						+ (light.y - newPoint.y) * abc.y
						+ (light.z - newPoint.z) * abc.z;
					if (colRatio < 0)
						colRatio = 0;
					out = colors[j];
					skip = j;
					coled = true;
				}
			}
		}
		bool shadow = false;
		if (coled) {
			float3 pos = newPoint;
			float3 dir;
			dir.x = light.x - pos.x;
			dir.y = light.y - pos.y;
			dir.z = light.z - pos.z;
			float dirLen = norm3df(dir.x, dir.y, dir.z);
			dir.x /= abs(dirLen);
			dir.y /= abs(dirLen);
			dir.z /= abs(dirLen);
			for (int id = 0; id < CountOfTrns; id++) {
				if (id == skip)
					continue;
				float3 nor = normals[id];
				//float upper = dRatios[j] + abc.x * xyz.x + abc.y * xyz.y + abc.z * xyz.z;
				//float lower = abc.x * dir.x + abc.y * dir.y + abc.z * dir.z;
				float u = dRatios[id] + nor.x * pos.x + nor.y * pos.y + nor.z * pos.z;
				float l = nor.x * dir.x + nor.y * dir.y + nor.z * dir.z;
				if (l == 0 && u >= 0)
					continue;
				float t = -(u / l);
				if (t < 0)
					continue;
				float3 sdPt;
				sdPt.x = dir.x * t + pos.x;
				sdPt.y = dir.y * t + pos.y;
				sdPt.z = dir.z * t + pos.z;
				if (nor.x != 0) {
					pts[0].x = sdPt.y;
					pts[0].y = sdPt.z;
					pts[1].x = trPoint1[id].y;
					pts[1].y = trPoint1[id].z;
					pts[2].x = trPoint2[id].y;
					pts[2].y = trPoint2[id].z;
					pts[3].x = trPoint3[id].y;
					pts[3].y = trPoint3[id].z;
				}
				else if (nor.y != 0) {
					pts[0].x = sdPt.x;
					pts[0].y = sdPt.z;
					pts[1].x = trPoint1[id].x;
					pts[1].y = trPoint1[id].z;
					pts[2].x = trPoint2[id].x;
					pts[2].y = trPoint2[id].z;
					pts[3].x = trPoint3[id].x;
					pts[3].y = trPoint3[id].z;
				}
				else {
					pts[0].x = sdPt.x;
					pts[0].y = sdPt.y;
					pts[1].x = trPoint1[id].x;
					pts[1].y = trPoint1[id].y;
					pts[2].x = trPoint2[id].x;
					pts[2].y = trPoint2[id].y;
					pts[3].x = trPoint3[id].x;
					pts[3].y = trPoint3[id].y;
				}
				float a;
				a = (pts[1].x - pts[0].x) * (pts[2].y - pts[1].y) - (pts[2].x - pts[1].x) * (pts[1].y - pts[0].y);
				float b;
				b = (pts[2].x - pts[0].x) * (pts[3].y - pts[2].y) - (pts[3].x - pts[2].x) * (pts[2].y - pts[0].y);
				float c;
				c = (pts[3].x - pts[0].x) * (pts[1].y - pts[3].y) - (pts[1].x - pts[3].x) * (pts[3].y - pts[0].y);
				bool isIn = (a >= 0 && b >= 0 && c >= 0) || (a <= 0 && b <= 0 && c <= 0);
				if (isIn) {
					shadow = true;
					break;
				}

			}
		}
		if (shadow) {
			colRatio = 1;
			out = y;
		}
		colRatio = pow(abs(colRatio), 0.3);
		out.x *= colRatio;
		out.y *= colRatio;
		out.z *= colRatio;
		outColors[i] = out;

	}
}
