
#include <hip/hip_runtime.h>
﻿extern "C" {

	__constant__ int CountOfTrns = 512;
	__constant__ float3 trPoint1[512];
	__constant__ float3 trPoint2[512];
	__constant__ float3 trPoint3[512];
	__constant__ float3 normals[512];
	__constant__ float dRatios[512];
	__constant__ float3 colors[512];
	__global__ void resultPixel(const float3* rays, const float3 xyz, const int count,const float3 light, float3* outColors) {
		int i = blockDim.x * blockIdx.x + threadIdx.x;
		if (i < count) {
			float3 mpl = rays[i];
			float3 baseColor;
			baseColor.x = 25;
			baseColor.y = 25;
			baseColor.z = 25;
			float3 red;
			red.x = 255;
			float3 green;
			green.y = 255;
			float3 blue;
			blue.z = 255;
			float minDist;
			minDist = 10000000;
			float3 out;
			out = baseColor;
			float3 newPoint;
			int skip;
			bool coled;
			for (int j = 0; j < CountOfTrns; j++)
			{
				float3 abc = normals[j];
				float upper = dRatios[j] + abc.x * xyz.x + abc.y * xyz.y + abc.z * xyz.z;
				float lower = abc.x * mpl.x + abc.y * mpl.y + abc.z * mpl.z;
				if (lower == 0 && upper == 0) {
					continue;
				}
				if (upper > 0 && lower == 0) {
					continue;
				}
				float t = -(upper / lower);
				if (t < 0) {
					continue;
				}
				newPoint.x = mpl.x * t;
				newPoint.y = mpl.y * t;
				newPoint.z = mpl.z * t;
				newPoint.x += xyz.x;
				newPoint.y += xyz.y;
				newPoint.z += xyz.z;
				float2 pts[4];
				if (abc.x != 0) {
					pts[0].x = newPoint.y;
					pts[0].y = newPoint.z;
					pts[1].x = trPoint1[j].y;
					pts[1].y = trPoint1[j].z;
					pts[2].x = trPoint2[j].y;
					pts[2].y = trPoint2[j].z;
					pts[3].x = trPoint3[j].y;
					pts[3].y = trPoint3[j].z;
				}
				else if (abc.y != 0) {
					pts[0].x = newPoint.x;
					pts[0].y = newPoint.z;
					pts[1].x = trPoint1[j].x;
					pts[1].y = trPoint1[j].z;
					pts[2].x = trPoint2[j].x;
					pts[2].y = trPoint2[j].z;
					pts[3].x = trPoint3[j].x;
					pts[3].y = trPoint3[j].z;
				}
				else {
					pts[0].x = newPoint.x;
					pts[0].y = newPoint.y;
					pts[1].x = trPoint1[j].x;
					pts[1].y = trPoint1[j].y;
					pts[2].x = trPoint2[j].x;
					pts[2].y = trPoint2[j].y;
					pts[3].x = trPoint3[j].x;
					pts[3].y = trPoint3[j].y;
				}
				float a;
				a = (pts[1].x - pts[0].x) * (pts[2].y - pts[1].y) - (pts[2].x - pts[1].x) * (pts[1].y - pts[0].y);
				float b;
				b = (pts[2].x - pts[0].x) * (pts[3].y - pts[2].y) - (pts[3].x - pts[2].x) * (pts[2].y - pts[0].y);
				float c;
				c = (pts[3].x - pts[0].x) * (pts[1].y - pts[3].y) - (pts[1].x - pts[3].x) * (pts[3].y - pts[0].y);
				bool isIn = (a >= 0 && b >= 0 && c >= 0) || (a <= 0 && b <= 0 && c <= 0);
				if (isIn) {
					
					float newLen = sqrt(pow(newPoint.x - xyz.x, 2) + pow(newPoint.y - xyz.y, 2) + pow(newPoint.z - xyz.z, 2));
					bool around = newLen
						< minDist;
					if (around) {
						float ratio;
						ratio = (light.x - newPoint.x) * abc.x
						+(light.y - newPoint.y) * abc.y
						+ (light.z - newPoint.z) * abc.z;
					minDist = newLen;
					ratio = pow(ratio, 0.3);
					out = colors[j];
					out.x *= ratio;
					out.y *= ratio;
					out.z *= ratio;
					skip = j;
					coled = true;
					}
				}				
			}
			if (coled) {
				float3 colxyz = newPoint;
				float3 tol;
				tol.x = light.x - colxyz.x;
				tol.y = light.y - colxyz.y;
				tol.z = light.z - colxyz.z;
				float len = sqrt(pow(tol.x, 2) + pow(tol.y, 2) + pow(tol.z, 2));
				tol.x /= len;
				tol.y /= len;
				tol.z /= len;
				bool shadow = false;
				for (int j = 0; j < CountOfTrns; j++)
				{
					if (j == skip)
						continue;
					float3 abc = normals[j];
					float upper = dRatios[j] + abc.x * colxyz.x + abc.y * colxyz.y + abc.z * colxyz.z;
					float lower = abc.x * tol.x + abc.y * tol.y + abc.z * tol.z;
					if (lower == 0 && upper == 0) {
						continue;
					}
					if (upper > 0 && lower == 0) {
						continue;
					}
					float t = -(upper / lower);
					if (t < 0) {
						continue;
					}
					float3 newPoint;
					newPoint.x = tol.x * t;
					newPoint.y = tol.y * t;
					newPoint.z = tol.z * t;
					newPoint.x += colxyz.x;
					newPoint.y += colxyz.y;
					newPoint.z += colxyz.z;
					float2 pts[4];
					if (abc.x != 0) {
						pts[0].x = newPoint.y;
						pts[0].y = newPoint.z;
						pts[1].x = trPoint1[j].y;
						pts[1].y = trPoint1[j].z;
						pts[2].x = trPoint2[j].y;
						pts[2].y = trPoint2[j].z;
						pts[3].x = trPoint3[j].y;
						pts[3].y = trPoint3[j].z;
					}
					else if (abc.y != 0) {
						pts[0].x = newPoint.x;
						pts[0].y = newPoint.z;
						pts[1].x = trPoint1[j].x;
						pts[1].y = trPoint1[j].z;
						pts[2].x = trPoint2[j].x;
						pts[2].y = trPoint2[j].z;
						pts[3].x = trPoint3[j].x;
						pts[3].y = trPoint3[j].z;
					}
					else {
						pts[0].x = newPoint.x;
						pts[0].y = newPoint.y;
						pts[1].x = trPoint1[j].x;
						pts[1].y = trPoint1[j].y;
						pts[2].x = trPoint2[j].x;
						pts[2].y = trPoint2[j].y;
						pts[3].x = trPoint3[j].x;
						pts[3].y = trPoint3[j].y;
					}
					float a;
					a = (pts[1].x - pts[0].x) * (pts[2].y - pts[1].y) - (pts[2].x - pts[1].x) * (pts[1].y - pts[0].y);
					float b;
					b = (pts[2].x - pts[0].x) * (pts[3].y - pts[2].y) - (pts[3].x - pts[2].x) * (pts[2].y - pts[0].y);
					float c;
					c = (pts[3].x - pts[0].x) * (pts[1].y - pts[3].y) - (pts[1].x - pts[3].x) * (pts[3].y - pts[0].y);
					bool isIn = (a >= 0 && b >= 0 && c >= 0) || (a <= 0 && b <= 0 && c <= 0);
					if (isIn) {
						float newLen = sqrt(pow(newPoint.x - colxyz.x, 2) + pow(newPoint.y - colxyz.y, 2) + pow(newPoint.z - colxyz.z, 2));
						bool around = newLen < minDist;
						if (around) {
							shadow = true;
							minDist = newLen;
						}
					}
				}
				if (shadow) {
					out.x = 0;
					out.y = 0;
					out.z = 0;
				}
			}
			outColors[i] = out;
		}
	}
}
