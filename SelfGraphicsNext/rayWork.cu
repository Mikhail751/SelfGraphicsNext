
#include <hip/hip_runtime.h>
﻿extern "C" {
	typedef struct {
		int objId;
		float3 p1;
		float3 p2;
		float3 p3;
		float3 nor;
		float3 col;
		float d;
	} PolygonCUDA;
	__constant__ float ToRad = 0.017453292519943295769236907684886;
	__constant__ float ToDeg = 57.295779513082320876798154814105;
	__global__ void resultPixel(const PolygonCUDA* pgs, const int polCount,
		const int2 resolution, const float3 xyz,
		const float3 light, const float fow,
		const float2 view, float3* outColors) {
		//Getting position of thread
		int row = blockIdx.y * blockDim.y + threadIdx.y;
		int col = blockIdx.x * blockDim.x + threadIdx.x;
		int i = col + row * resolution.x;
		if (col >= resolution.x || row >= resolution.y) {
			return;
		}
		//Init variables
		float3 dir;
		float3 baseColor;
		float3 out;
		float3 newPoint;
		float3 colPoint;
		float3 abc;
		float2 temp;
		float2 pts[4];
		float2 tg;
		int2 halfRes;
		float minDist = -1;
		float colRatio = 1;
		float upper;
		float lower;
		float a, b, c;
		bool coled = false;
		int skip;
		PolygonCUDA pol;
		float radValue;

		//Setting values
		baseColor.x = 16;
		baseColor.y = 16;
		baseColor.z = 16;
		out = baseColor;

		//Getting current thead directon
		dir.y = col;
		dir.z = row;
		halfRes.x = resolution.x / 2;
		halfRes.y = resolution.y / 2;
		dir.y -= halfRes.x;
		dir.z -= halfRes.y;
		dir.z *= -1;
		dir.y /= halfRes.x;
		dir.z /= halfRes.y;
		float halfLenHor = sinf(fow * ToRad / 2);
		dir.y *= halfLenHor;
		dir.x = sqrt(1 - powf(dir.y, 2));
		float halfLenVer = halfLenHor / resolution.x * resolution.y;
		dir.z *= halfLenVer;
		float verRatio = sqrt(1 - pow(dir.z, 2));
		if (view.y != 0) {
			radValue = view.y * ToRad;
			tg.x = cos(radValue);
			tg.y = sin(radValue);
			temp.y = verRatio * tg.x - dir.z * tg.y;
			temp.x = verRatio * tg.y + dir.z * tg.x;
			dir.z = temp.x;
			verRatio = temp.y;
		}
		dir.x *= verRatio;
		dir.y *= verRatio;
		if (view.x != 0) {
			radValue = view.x * ToRad;
			tg.x = cos(radValue);
			tg.y = sin(radValue);
			temp.x = dir.x * tg.x - dir.y * tg.y;
			temp.y = dir.x * tg.y + dir.y * tg.x;
			dir.x = temp.x;
			dir.y = temp.y;
		}

		//Counting first colision point
		for (int j = 0; j < polCount; j++)
		{
			pol = pgs[j];
			abc = pol.nor;
			upper = pol.d + abc.x * xyz.x + abc.y * xyz.y + abc.z * xyz.z;
			lower = abc.x * dir.x + abc.y * dir.y + abc.z * dir.z;
			if (lower == 0 && upper == 0) {
				continue;
			}
			if (upper > 0 && lower == 0) {
				continue;
			}
			float t = -(upper / lower);
			if (t < 0) {
				continue;
			}
			newPoint.x = dir.x * t + xyz.x;
			newPoint.y = dir.y * t + xyz.y;
			newPoint.z = dir.z * t + xyz.z;
			if (abc.x != 0) {
				pts[0].x = newPoint.y;
				pts[0].y = newPoint.z;
				pts[1].x = pol.p1.y;
				pts[1].y = pol.p1.z;
				pts[2].x = pol.p2.y;
				pts[2].y = pol.p2.z;
				pts[3].x = pol.p3.y;
				pts[3].y = pol.p3.z;
			}
			else if (abc.y != 0) {
				pts[0].x = newPoint.x;
				pts[0].y = newPoint.z;
				pts[1].x = pol.p1.x;
				pts[1].y = pol.p1.z;
				pts[2].x = pol.p2.x;
				pts[2].y = pol.p2.z;
				pts[3].x = pol.p3.x;
				pts[3].y = pol.p3.z;
			}
			else {
				pts[0].x = newPoint.x;
				pts[0].y = newPoint.y;
				pts[1].x = pol.p1.x;
				pts[1].y = pol.p1.y;
				pts[2].x = pol.p2.x;
				pts[2].y = pol.p2.y;
				pts[3].x = pol.p3.x;
				pts[3].y = pol.p3.y;
			}
			a = (pts[1].x - pts[0].x) * (pts[2].y - pts[1].y) - (pts[2].x - pts[1].x) * (pts[1].y - pts[0].y);
			b = (pts[2].x - pts[0].x) * (pts[3].y - pts[2].y) - (pts[3].x - pts[2].x) * (pts[2].y - pts[0].y);
			c = (pts[3].x - pts[0].x) * (pts[1].y - pts[3].y) - (pts[1].x - pts[3].x) * (pts[3].y - pts[0].y);
			bool isIn = (a >= 0 && b >= 0 && c >= 0) || (a <= 0 && b <= 0 && c <= 0);
			if (isIn) {

				float newLen = sqrt(pow(newPoint.x - xyz.x, 2) + pow(newPoint.y - xyz.y, 2) + pow(newPoint.z - xyz.z, 2));
				bool around;
				if (minDist == -1) {
					minDist = newLen;
					around = true;
				}
				else {
					around = newLen < minDist;
				}
				if (around) {
					minDist = newLen;
					colRatio = (light.x - newPoint.x) * abc.x
						+ (light.y - newPoint.y) * abc.y
						+ (light.z - newPoint.z) * abc.z;
					if (colRatio < 0)
						colRatio = 0;
					out = pol.col;
					skip = j;
					coled = true;
					colPoint = newPoint;
				}
			}
		}
		bool shadow = false;
		if (coled) {
			float3 pos = colPoint;
			dir.x = light.x - pos.x;
			dir.y = light.y - pos.y;
			dir.z = light.z - pos.z;
			float dirLen = norm3df(dir.x, dir.y, dir.z);
			dir.x /= dirLen;
			dir.y /= dirLen;
			dir.z /= dirLen;
			for (int id = 0; id < polCount; id++) {
				if (id == skip)
					continue;
				pol = pgs[id];
				float3 nor = pol.nor;
				float u = pol.d + nor.x * pos.x + nor.y * pos.y + nor.z * pos.z;
				float l = nor.x * dir.x + nor.y * dir.y + nor.z * dir.z;
				if (l == 0 && u >= 0)
					continue;
				float t = -(u / l);
				if (t < 0)
					continue;
				float3 sdPt;
				sdPt.x = dir.x * t + pos.x;
				sdPt.y = dir.y * t + pos.y;
				sdPt.z = dir.z * t + pos.z;
				if (nor.x != 0) {
					pts[0].x = sdPt.y;
					pts[0].y = sdPt.z;
					pts[1].x = pol.p1.y;
					pts[1].y = pol.p1.z;
					pts[2].x = pol.p2.y;
					pts[2].y = pol.p2.z;
					pts[3].x = pol.p3.y;
					pts[3].y = pol.p3.z;
				}
				else if (nor.y != 0) {
					pts[0].x = sdPt.x;
					pts[0].y = sdPt.z;
					pts[1].x = pol.p1.x;
					pts[1].y = pol.p1.z;
					pts[2].x = pol.p2.x;
					pts[2].y = pol.p2.z;
					pts[3].x = pol.p3.x;
					pts[3].y = pol.p3.z;
				}
				else {
					pts[0].x = sdPt.x;
					pts[0].y = sdPt.y;
					pts[1].x = pol.p1.x;
					pts[1].y = pol.p1.y;
					pts[2].x = pol.p2.x;
					pts[2].y = pol.p2.y;
					pts[3].x = pol.p3.x;
					pts[3].y = pol.p3.y;
				}
				a = (pts[1].x - pts[0].x) * (pts[2].y - pts[1].y) - (pts[2].x - pts[1].x) * (pts[1].y - pts[0].y);
				b = (pts[2].x - pts[0].x) * (pts[3].y - pts[2].y) - (pts[3].x - pts[2].x) * (pts[2].y - pts[0].y);
				c = (pts[3].x - pts[0].x) * (pts[1].y - pts[3].y) - (pts[1].x - pts[3].x) * (pts[3].y - pts[0].y);
				bool isIn = (a >= 0 && b >= 0 && c >= 0) || (a <= 0 && b <= 0 && c <= 0);
				if (isIn) {
					shadow = true;
					break;
				}

			}
		}
		if (shadow) {
			colRatio *= 0.25;
		}
		colRatio = pow(abs(colRatio), 0.3);
		out.x *= colRatio;
		out.y *= colRatio;
		out.z *= colRatio;
		outColors[i] = out;

	}
}
