
#include <hip/hip_runtime.h>
﻿extern "C" {
	typedef struct {
		int objId;
		float3 p1;
		float3 p2;
		float3 p3;
		float3 nor;
		float3 col;
		float d;
	} PolygonCUDA;
	__constant__ float ToRad = 0.017453292519943295769236907684886;
	__constant__ float ToDeg = 57.295779513082320876798154814105;
	__global__ void resultPixel(const PolygonCUDA* pgs, const int polCount,
		const int2 resolution, const float3 xyz,
		const float3 light, const float fow,
		const float2 view, float3* outColors) {
		//Getting position of thread
		int row = blockIdx.y * blockDim.y + threadIdx.y;
		int col = blockIdx.x * blockDim.x + threadIdx.x;
		int i = col + row * resolution.x;
		if (col >= resolution.x || row >= resolution.y) {
			return;
		}
		//Init variables
		float3 dir;
		float3 baseColor;
		float3 out;
		float3 newPoint;
		float3 colPoint;
		float3 abc;
		float3 tol;

		float2 temp;
		float2 pts[4];
		float2 tg;
		int2 halfRes;

		float tolLen;
		float minDist = -1;
		float colRatio = 1;
		float upper;
		float lower;
		float a, b, c;
		bool coled = false;
		int skip;
		PolygonCUDA pol;
		float radValue;

		//Setting values
		baseColor.x = 16;
		baseColor.y = 16;
		baseColor.z = 16;
		out = baseColor;

		//Getting current thead directon
		dir.y = col;
		dir.z = row;
		halfRes.x = resolution.x / 2;
		halfRes.y = resolution.y / 2;
		dir.y -= halfRes.x;
		dir.z -= halfRes.y;
		dir.z *= -1;
		dir.y /= halfRes.x;
		dir.z /= halfRes.y;
		float halfLenHor = sinf(fow * ToRad / 2);
		dir.y *= halfLenHor;
		dir.x = sqrt(1 - powf(dir.y, 2));
		float halfLenVer = halfLenHor / resolution.x * resolution.y;
		dir.z *= halfLenVer;
		float verRatio = sqrt(1 - pow(dir.z, 2));
		if (view.y != 0) {
			radValue = view.y * ToRad;
			tg.x = cos(radValue);
			tg.y = sin(radValue);
			temp.y = verRatio * tg.x - dir.z * tg.y;
			temp.x = verRatio * tg.y + dir.z * tg.x;
			dir.z = temp.x;
			verRatio = temp.y;
		}
		dir.x *= verRatio;
		dir.y *= verRatio;
		if (view.x != 0) {
			radValue = view.x * ToRad;
			tg.x = cos(radValue);
			tg.y = sin(radValue);
			temp.x = dir.x * tg.x - dir.y * tg.y;
			temp.y = dir.x * tg.y + dir.y * tg.x;
			dir.x = temp.x;
			dir.y = temp.y;
		}



		for (int j = 0; j < polCount; j++)
		{
			pol = pgs[j];
			abc = pol.nor;
			upper = pol.d + abc.x * xyz.x + abc.y * xyz.y + abc.z * xyz.z;
			lower = abc.x * dir.x + abc.y * dir.y + abc.z * dir.z;
			if (lower == 0 && upper == 0) {
				continue;
			}
			if (upper > 0 && lower == 0) {
				continue;
			}
			float t = -(upper / lower);
			if (t < 0) {
				continue;
			}
			newPoint.x = dir.x * t + xyz.x;
			newPoint.y = dir.y * t + xyz.y;
			newPoint.z = dir.z * t + xyz.z;
			if (abc.x != 0) {
				pts[0].x = newPoint.y;
				pts[0].y = newPoint.z;
				pts[1].x = pol.p1.y;
				pts[1].y = pol.p1.z;
				pts[2].x = pol.p2.y;
				pts[2].y = pol.p2.z;
				pts[3].x = pol.p3.y;
				pts[3].y = pol.p3.z;
			}
			else if (abc.y != 0) {
				pts[0].x = newPoint.x;
				pts[0].y = newPoint.z;
				pts[1].x = pol.p1.x;
				pts[1].y = pol.p1.z;
				pts[2].x = pol.p2.x;
				pts[2].y = pol.p2.z;
				pts[3].x = pol.p3.x;
				pts[3].y = pol.p3.z;
			}
			else {
				pts[0].x = newPoint.x;
				pts[0].y = newPoint.y;
				pts[1].x = pol.p1.x;
				pts[1].y = pol.p1.y;
				pts[2].x = pol.p2.x;
				pts[2].y = pol.p2.y;
				pts[3].x = pol.p3.x;
				pts[3].y = pol.p3.y;
			}
			a = (pts[1].x - pts[0].x) * (pts[2].y - pts[1].y) - (pts[2].x - pts[1].x) * (pts[1].y - pts[0].y);
			b = (pts[2].x - pts[0].x) * (pts[3].y - pts[2].y) - (pts[3].x - pts[2].x) * (pts[2].y - pts[0].y);
			c = (pts[3].x - pts[0].x) * (pts[1].y - pts[3].y) - (pts[1].x - pts[3].x) * (pts[3].y - pts[0].y);
			bool isIn = (a >= 0 && b >= 0 && c >= 0) || (a <= 0 && b <= 0 && c <= 0);
			if (isIn) {

				float newLen = sqrt(pow(newPoint.x - xyz.x, 2) + pow(newPoint.y - xyz.y, 2) + pow(newPoint.z - xyz.z, 2));
				bool around;
				if (minDist == -1) {
					minDist = newLen;
					around = true;
				}
				else {
					around = newLen < minDist;
				}
				if (around) {
					minDist = newLen;
					tol.x = light.x - newPoint.x;
					tol.y = light.y - newPoint.y;
					tol.z = light.z - newPoint.z;
					colRatio = tol.x * abc.x
						+ tol.y * abc.y
						+ tol.z * abc.z;
					tolLen = norm3df(tol.x, tol.y, tol.z);
					colRatio /= tolLen;
					if (colRatio < 0)
						colRatio = 0;
					out = pol.col;
					skip = j;
					coled = true;
					colPoint = newPoint;
				}
			}
		}
		bool shadow = false;
		if (coled) {
			float3 pos = colPoint;
			dir.x = light.x - pos.x;
			dir.y = light.y - pos.y;
			dir.z = light.z - pos.z;
			float dirLen = norm3df(dir.x, dir.y, dir.z);
			dir.x /= dirLen;
			dir.y /= dirLen;
			dir.z /= dirLen;
			for (int id = 0; id < polCount; id++) {
				if (id == skip)
					continue;
				pol = pgs[id];
				float3 nor = pol.nor;
				float u = pol.d + nor.x * pos.x + nor.y * pos.y + nor.z * pos.z;
				float l = nor.x * dir.x + nor.y * dir.y + nor.z * dir.z;
				if (l == 0 && u >= 0)
					continue;
				float t = -(u / l);
				if (t < 0 || t > dirLen)
					continue;
				float3 sdPt;
				sdPt.x = dir.x * t + pos.x;
				sdPt.y = dir.y * t + pos.y;
				sdPt.z = dir.z * t + pos.z;
				if (nor.x != 0) {
					pts[0].x = sdPt.y;
					pts[0].y = sdPt.z;
					pts[1].x = pol.p1.y;
					pts[1].y = pol.p1.z;
					pts[2].x = pol.p2.y;
					pts[2].y = pol.p2.z;
					pts[3].x = pol.p3.y;
					pts[3].y = pol.p3.z;
				}
				else if (nor.y != 0) {
					pts[0].x = sdPt.x;
					pts[0].y = sdPt.z;
					pts[1].x = pol.p1.x;
					pts[1].y = pol.p1.z;
					pts[2].x = pol.p2.x;
					pts[2].y = pol.p2.z;
					pts[3].x = pol.p3.x;
					pts[3].y = pol.p3.z;
				}
				else {
					pts[0].x = sdPt.x;
					pts[0].y = sdPt.y;
					pts[1].x = pol.p1.x;
					pts[1].y = pol.p1.y;
					pts[2].x = pol.p2.x;
					pts[2].y = pol.p2.y;
					pts[3].x = pol.p3.x;
					pts[3].y = pol.p3.y;
				}
				a = (pts[1].x - pts[0].x) * (pts[2].y - pts[1].y) - (pts[2].x - pts[1].x) * (pts[1].y - pts[0].y);
				b = (pts[2].x - pts[0].x) * (pts[3].y - pts[2].y) - (pts[3].x - pts[2].x) * (pts[2].y - pts[0].y);
				c = (pts[3].x - pts[0].x) * (pts[1].y - pts[3].y) - (pts[1].x - pts[3].x) * (pts[3].y - pts[0].y);
				bool isIn = (a >= 0 && b >= 0 && c >= 0) || (a <= 0 && b <= 0 && c <= 0);
				if (isIn) {
					shadow = true;
					break;
				}

			}
		}
		if (shadow) {

			colRatio *= 0.1;
		}
		/*float3 toNewp;
		toNewp.x = colPoint.x - xyz.x;
		toNewp.y = colPoint.y - xyz.y;
		toNewp.z = colPoint.z - xyz.z;
		tolLen = norm3df(tol.x, tol.y, tol.z);

		if (norm3df(toNewp.x, toNewp.y, toNewp.z) < tolLen) {
			float scTol = tol.x * dir.x + tol.y * dir.y + tol.z * dir.z;
			scTol /= tolLen;
			if (scTol > 0.999) {
				out.x = 255;
				out.y = 255;
				out.z = 125;
				colRatio = 1;
			}
		}*/



		//Counting first colision point
		if (norm)
			colRatio = pow(abs(colRatio), 0.22);
		out.x *= colRatio;
		out.y *= colRatio;
		out.z *= colRatio;
		outColors[i] = out;

	}
}
