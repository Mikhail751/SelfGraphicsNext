
#include <hip/hip_runtime.h>
﻿extern "C" {
	__constant__ int CountOfTrns = 512;
	__constant__ float3 trPoint1[512];
	__constant__ float3 trPoint2[512];
	__constant__ float3 trPoint3[512];
	__constant__ float3 normals[512];
	__constant__ float dRatios[512];
	__constant__ float3 colors[512];
	__global__ void resultPixel(const float3* rays, const float3 xyz,const int count, float3* outColors) {
		int i = blockDim.x * blockIdx.x + threadIdx.x;
		if (i < count) {
			float3 mpl = rays[i];
			float3 baseColor;
			baseColor.x = 25;
			baseColor.y = 25;
			baseColor.z = 25;
			float3 red;
			red.x = 255;
			float3 green;
			green.y = 255;
			float3 blue;
			blue.z = 255;
			//outColors[i] = baseColor;
			float minDist;
			minDist = 512000000;
			float3 out;
			//out = base
			for (int j = 0; j < CountOfTrns; j++)
			{
				float3 abc = normals[j];
				float upper = dRatios[j] + abc.x * xyz.x + abc.y * xyz.y + abc.z * xyz.z;
				float lower = abc.x * mpl.x + abc.y * mpl.y + abc.z * mpl.z;
				if (lower == 0 && upper == 0) {
				}
				else if (upper > 0 && lower == 0) {
				}
				else {
					float t = -(upper / lower);
					if (t < 0) {
					}
					else {
						float3 newPoint;
						newPoint.x = mpl.x * t;
						newPoint.y = mpl.y * t;
						newPoint.z = mpl.z * t;
						float newLen = sqrt(pow(newPoint.x, 2) + pow(newPoint.y, 2) + pow(newPoint.z, 2));
						bool around = newLen < minDist;
						if (around) {
							newPoint.x += xyz.x;
							newPoint.y += xyz.y;
							newPoint.z += xyz.z;
							float2 pts[4];
							if (abc.x != 0) {
								pts[0].x = newPoint.y;
								pts[0].y = newPoint.z;
								pts[1].x = trPoint1[j].y;
								pts[1].y = trPoint1[j].z;
								pts[2].x = trPoint2[j].y;
								pts[2].y = trPoint2[j].z;
								pts[3].x = trPoint3[j].y;
								pts[3].y = trPoint3[j].z;
							}
							else if (abc.y != 0) {
								pts[0].x = newPoint.x;
								pts[0].y = newPoint.z;
								pts[1].x = trPoint1[j].x;
								pts[1].y = trPoint1[j].z;
								pts[2].x = trPoint2[j].x;
								pts[2].y = trPoint2[j].z;
								pts[3].x = trPoint3[j].x;
								pts[3].y = trPoint3[j].z;
							}
							else {
								pts[0].x = newPoint.x;
								pts[0].y = newPoint.y;
								pts[1].x = trPoint1[j].x;
								pts[1].y = trPoint1[j].y;
								pts[2].x = trPoint2[j].x;
								pts[2].y = trPoint2[j].y;
								pts[3].x = trPoint3[j].x;
								pts[3].y = trPoint3[j].y;
							}
							float a;
							a = (pts[1].x - pts[0].x) * (pts[2].y - pts[1].y) - (pts[2].x - pts[1].x) * (pts[1].y - pts[0].y);
							float b;
							b = (pts[2].x - pts[0].x) * (pts[3].y - pts[2].y) - (pts[3].x - pts[2].x) * (pts[2].y - pts[0].y);
							float c;
							c = (pts[3].x - pts[0].x) * (pts[1].y - pts[3].y) - (pts[1].x - pts[3].x) * (pts[3].y - pts[0].y);
							bool isIn = (a >= 0 && b >= 0 && c >= 0) || (a <= 0 && b <= 0 && c <= 0);
							if (isIn) {
								minDist = newLen;
								out = red;
							}
							else {
								out = green;
							}
						}
					}
				}
			}
			outColors[i] = out;
		}
	}
}