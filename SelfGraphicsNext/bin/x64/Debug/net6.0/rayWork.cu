
#include <hip/hip_runtime.h>
﻿extern "C" {

	__constant__ int CountOfTrns = 512;
	__constant__ float3 trPoint1[512];
	__constant__ float3 trPoint2[512];
	__constant__ float3 trPoint3[512];
	__constant__ float3 normals[512];
	__constant__ float dRatios[512];
	__constant__ float3 colors[512];
	__global__ void resultPixel(const float3* rays, const float3 xyz, const int count, const float3 light, float3* outColors) {
		int i = blockDim.x * blockIdx.x + threadIdx.x;
		if (i < count) {
			float3 mpl = rays[i];
			float3 baseColor;
			baseColor.x = 16;
			baseColor.y = 16;
			baseColor.z = 16;
			float minDist = 10000000;
			float3 out;
			out = baseColor;
			float3 newPoint;
			int skip;
			bool coled = false;
			float colRatio = 1;
			float2 pts[4];
			for (int j = 0; j < CountOfTrns; j++)
			{
				float3 abc = normals[j];
				float upper = dRatios[j] + abc.x * xyz.x + abc.y * xyz.y + abc.z * xyz.z;
				float lower = abc.x * mpl.x + abc.y * mpl.y + abc.z * mpl.z;
				if (lower == 0 && upper == 0) {
					continue;
				}
				if (upper > 0 && lower == 0) {
					continue;
				}
				float t = -(upper / lower);
				if (t < 0) {
					continue;
				}
				newPoint.x = mpl.x * t + xyz.x;
				newPoint.y = mpl.y * t + xyz.y;
				newPoint.z = mpl.z * t + xyz.z;
				if (abc.x != 0) {
					pts[0].x = newPoint.y;
					pts[0].y = newPoint.z;
					pts[1].x = trPoint1[j].y;
					pts[1].y = trPoint1[j].z;
					pts[2].x = trPoint2[j].y;
					pts[2].y = trPoint2[j].z;
					pts[3].x = trPoint3[j].y;
					pts[3].y = trPoint3[j].z;
				}
				else if (abc.y != 0) {
					pts[0].x = newPoint.x;
					pts[0].y = newPoint.z;
					pts[1].x = trPoint1[j].x;
					pts[1].y = trPoint1[j].z;
					pts[2].x = trPoint2[j].x;
					pts[2].y = trPoint2[j].z;
					pts[3].x = trPoint3[j].x;
					pts[3].y = trPoint3[j].z;
				}
				else {
					pts[0].x = newPoint.x;
					pts[0].y = newPoint.y;
					pts[1].x = trPoint1[j].x;
					pts[1].y = trPoint1[j].y;
					pts[2].x = trPoint2[j].x;
					pts[2].y = trPoint2[j].y;
					pts[3].x = trPoint3[j].x;
					pts[3].y = trPoint3[j].y;
				}
				float a;
				a = (pts[1].x - pts[0].x) * (pts[2].y - pts[1].y) - (pts[2].x - pts[1].x) * (pts[1].y - pts[0].y);
				float b;
				b = (pts[2].x - pts[0].x) * (pts[3].y - pts[2].y) - (pts[3].x - pts[2].x) * (pts[2].y - pts[0].y);
				float c;
				c = (pts[3].x - pts[0].x) * (pts[1].y - pts[3].y) - (pts[1].x - pts[3].x) * (pts[3].y - pts[0].y);
				bool isIn = (a >= 0 && b >= 0 && c >= 0) || (a <= 0 && b <= 0 && c <= 0);
				if (isIn) {

					float newLen = sqrt(pow(newPoint.x - xyz.x, 2) + pow(newPoint.y - xyz.y, 2) + pow(newPoint.z - xyz.z, 2));
					bool around = newLen < minDist;
					if (around) {
						minDist = newLen;
						colRatio = (light.x - newPoint.x) * abc.x
							+ (light.y - newPoint.y) * abc.y
							+ (light.z - newPoint.z) * abc.z;
						if (colRatio < 0)
							colRatio = 0;
						out = colors[j];
						skip = j;
						coled = true;
					}
				}
			}
			bool shadow = false;
			if (coled) {
				float3 pos = newPoint;
				float3 dir;
				dir.x = light.x - pos.x;
				dir.y = light.y - pos.y;
				dir.z = light.z - pos.z;
				float dirLen = norm3df(dir.x, dir.y, dir.z);
				dir.x /= abs(dirLen);
				dir.y /= abs(dirLen);
				dir.z /= abs(dirLen);
				for (int id = 0; id < CountOfTrns; id++) {
					if (id == skip)
						continue;
					float3 nor = normals[id];
					//float upper = dRatios[j] + abc.x * xyz.x + abc.y * xyz.y + abc.z * xyz.z;
					//float lower = abc.x * mpl.x + abc.y * mpl.y + abc.z * mpl.z;
					float u = dRatios[id] + nor.x * pos.x + nor.y * pos.y + nor.z * pos.z;
					float l = nor.x * dir.x + nor.y * dir.y + nor.z * dir.z;
					if (l == 0 && u >= 0)
						continue;
					float t = -(u / l);
					if (t < 0)
						continue;
					float3 sdPt;
					sdPt.x = dir.x * t + pos.x;
					sdPt.y = dir.y * t + pos.y;
					sdPt.z = dir.z * t + pos.z;
					if (nor.x != 0) {
						pts[0].x = sdPt.y;
						pts[0].y = sdPt.z;
						pts[1].x = trPoint1[id].y;
						pts[1].y = trPoint1[id].z;
						pts[2].x = trPoint2[id].y;
						pts[2].y = trPoint2[id].z;
						pts[3].x = trPoint3[id].y;
						pts[3].y = trPoint3[id].z;
					}
					else if (nor.y != 0) {
						pts[0].x = sdPt.x;
						pts[0].y = sdPt.z;
						pts[1].x = trPoint1[id].x;
						pts[1].y = trPoint1[id].z;
						pts[2].x = trPoint2[id].x;
						pts[2].y = trPoint2[id].z;
						pts[3].x = trPoint3[id].x;
						pts[3].y = trPoint3[id].z;
					}
					else {
						pts[0].x = sdPt.x;
						pts[0].y = sdPt.y;
						pts[1].x = trPoint1[id].x;
						pts[1].y = trPoint1[id].y;
						pts[2].x = trPoint2[id].x;
						pts[2].y = trPoint2[id].y;
						pts[3].x = trPoint3[id].x;
						pts[3].y = trPoint3[id].y;
					}
					float a;
					a = (pts[1].x - pts[0].x) * (pts[2].y - pts[1].y) - (pts[2].x - pts[1].x) * (pts[1].y - pts[0].y);
					float b;
					b = (pts[2].x - pts[0].x) * (pts[3].y - pts[2].y) - (pts[3].x - pts[2].x) * (pts[2].y - pts[0].y);
					float c;
					c = (pts[3].x - pts[0].x) * (pts[1].y - pts[3].y) - (pts[1].x - pts[3].x) * (pts[3].y - pts[0].y);
					bool isIn = (a >= 0 && b >= 0 && c >= 0) || (a <= 0 && b <= 0 && c <= 0);
					if (isIn) {
						shadow = true;
						break;
					}

				}
			}
			if (shadow) {
				colRatio = 0;
			}
			colRatio = pow(abs(colRatio), 0.3);
			out.x *= colRatio;
			out.y *= colRatio;
			out.z *= colRatio;
			outColors[i] = out;
		}
	}
}
