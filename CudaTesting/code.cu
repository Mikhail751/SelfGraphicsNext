#include "hip/hip_runtime.h"
﻿extern "C" {
	__constant__ double ToRad = 0.017453292519943295769236907684886;
	__constant__ double ToDeg = 57.295779513082320876798154814105;
	typedef struct {
		float3 p1;
		float3 p2;
		float3 p3;
		float3 nor;
		float3 col;
		float d;
	} PolygonCUDA;
	__global__ void func(const PolygonCUDA* data, PolygonCUDA* out) {
		int2 baseRes;
		baseRes.x = blockDim.x * gridDim.x;
		baseRes.y = blockDim.y * gridDim.y;
		int col = blockIdx.y * blockDim.y + threadIdx.y;
		int row = blockIdx.x * blockDim.x + threadIdx.x;
		int i = col + row * baseRes.x;
		PolygonCUDA pc = data[i];
		out[i] = pc;
		//double3 id;
		//id.y = col;
		//id.z = row;
		//int2 halfRes;
		//halfRes.x = baseRes.x / 2;
		//halfRes.y = baseRes.y / 2;
		//id.y -= halfRes.x;
		//id.z -= halfRes.y;
		//id.z *= -1;
		//id.y /= halfRes.x;
		//id.z /= halfRes.y;
		//double halfLenHor = sin(fow * ToRad / 2);
		//id.y *= halfLenHor;
		//id.x = sqrt(1 - pow(id.y, 2));
		//double halfLenVer = halfLenHor / baseRes.x * baseRes.y;
		//id.z *= halfLenVer;
		//double verRatio = sqrt(1 - pow(id.z, 2));
		//if (view.y != 0) {
		//	double2 temp;
		//	temp.y = verRatio * cos(view.y * ToRad) - id.z * sin(view.y * ToRad);
		//	temp.x = verRatio * sin(view.y * ToRad) + id.z * cos(view.y * ToRad);
		//	id.z = temp.x;
		//	verRatio = temp.y;
		//}
		//id.x *= verRatio;
		//id.y *= verRatio;
		//if (view.x != 0) {
		//	double2 temp;
		//	temp.x = id.x * cos(view.x * ToRad) - id.y * sin(view.x * ToRad);
		//	temp.y = id.x * sin(view.x * ToRad) + id.y * cos(view.x * ToRad);
		//	id.x = temp.x;
		//	id.y = temp.y;
		//}
		testType tt;
		tt.a = col;
		tt.b = row;
		tt.c = col * row;
		out[i] = tt;;

	}
}