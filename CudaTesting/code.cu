
#include <hip/hip_runtime.h>
﻿extern "C" {
	__constant__ double ToRad = 0.017453292519943295769236907684886;
	__constant__ double ToDeg = 57.295779513082320876798154814105;
	__global__ void func(double3* out, const double fow, double2 view) {
		int2 baseRes;
		baseRes.x = blockDim.x * gridDim.x;
		baseRes.y = blockDim.y * gridDim.y;
		int col = blockIdx.y * blockDim.y + threadIdx.y;
		int row = blockIdx.x * blockDim.x + threadIdx.x;
		int i = col + row * baseRes.x;

		double3 id;
		id.y = col;
		id.z = row;
		int2 halfRes;
		halfRes.x = baseRes.x / 2;
		halfRes.y = baseRes.y / 2;
		id.y -= halfRes.x;
		id.z -= halfRes.y;
		id.z *= -1;
		id.y /= halfRes.x;
		id.z /= halfRes.y;
		double halfLenHor = sin(fow * ToRad / 2);
		id.y *= halfLenHor;
		id.x = sqrt(1 - pow(id.y, 2));
		double halfLenVer = halfLenHor / baseRes.x * baseRes.y;
		id.z *= halfLenVer;
		double verRatio = sqrt(1 - pow(id.z, 2));
		if (view.y != 0) {
			double2 temp;
			temp.y = verRatio * cos(view.y * ToRad) - id.z * sin(view.y * ToRad);
			temp.x = verRatio * sin(view.y * ToRad) + id.z * cos(view.y * ToRad);
			id.z = temp.x;
			verRatio = temp.y;
		}
		id.x *= verRatio;
		id.y *= verRatio;
		if (view.x != 0) {
			double2 temp;
			temp.x = id.x * cos(view.x * ToRad) - id.y * sin(view.x * ToRad);
			temp.y = id.x * sin(view.x * ToRad) + id.y * cos(view.x * ToRad);
			id.x = temp.x;
			id.y = temp.y;
		}
		out[i] = id;

	}
}